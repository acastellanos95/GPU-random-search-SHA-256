
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <iomanip>
#include <hiprand/hiprand_kernel.h>

__device__ uint32_t Ch(uint32_t x, uint32_t y, uint32_t z){
  return (x & y) ^ ((~x) & z);
}

__device__ uint32_t Maj(uint32_t x, uint32_t y, uint32_t z){
  return (x & y) ^ (x & z) ^ (y & z);
}

__device__ uint32_t ROTR(uint32_t x, unsigned long n){
  return (x >> n) | ( x << (32-n));
}

__device__ uint32_t SHR(uint32_t x, unsigned long n){
  return (x >> n);
}

__device__ uint32_t Sigma_0(uint32_t x){
  return ROTR(x, 2) ^ ROTR(x, 13) ^ ROTR(x, 22);
}

__device__ uint32_t Sigma_1(uint32_t x){
  return ROTR(x, 6) ^ ROTR(x, 11) ^ ROTR(x, 25);
}

__device__ uint32_t sigma_0(uint32_t x){
  return ROTR(x, 7) ^ ROTR(x, 18) ^ SHR(x, 3);
}

__device__ uint32_t sigma_1(uint32_t x){
  return ROTR(x, 17) ^ ROTR(x, 19) ^ SHR(x, 10);
}

__device__ __constant__ uint32_t constants[64] = {0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
                                                  0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
                                                  0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
                                                  0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
                                                  0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
                                                  0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
                                                  0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
                                                  0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

__device__ __constant__ uint32_t padding[14] = {0x80000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,
                                                0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000040};

__device__ __constant__ uint32_t masks[8] = {0x0000000F,0x000000FF,0x00000FFF,0x0000FFFF,0x000FFFFF,0x00FFFFFF,0x0FFFFFFF,0xFFFFFFFF};

__global__ void setup_kernel(hiprandState *state, unsigned long long int seed)
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  /* Each thread gets same seed (cryptographic quality), a different sequence
     number, no offset */
  hiprand_init(seed, id, 0, &state[id]);
}

// Sabemos que m tiene 2 indices y su padding ya está en el device
__global__ void SHA_256_64(hiprandState *state, uint32_t *hash, uint32_t *m_device) {

  // 64 bit message
  uint32_t m[2];
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  /* Copy state to local memory for efficiency */
  hiprandState localState = state[id];
  /* Generate pseudo-random unsigned ints */
  uint32_t h[8];
  uint32_t W[64];
  m[0] = hiprand(&localState);
  m[1] = hiprand(&localState);

  m_device[id * 2] = m[0];
  m_device[id * 2 + 1] = m[1];

  h[0] = 0x6a09e667;
  h[1] = 0xbb67ae85;
  h[2] = 0x3c6ef372;
  h[3] = 0xa54ff53a;
  h[4] = 0x510e527f;
  h[5] = 0x9b05688c;
  h[6] = 0x1f83d9ab;
  h[7] = 0x5be0cd19;

  // Message schedule
  W[0] = m[0];
  W[1] = m[1];
  W[2] = padding[0];
  W[3] = padding[1];
  W[4] = padding[2];
  W[5] = padding[3];
  W[6] = padding[4];
  W[7] = padding[5];
  W[8] = padding[6];
  W[9] = padding[7];
  W[10] = padding[8];
  W[11] = padding[9];
  W[12] = padding[10];
  W[13] = padding[11];
  W[14] = padding[12];
  W[15] = padding[13];

  for(int i = 16; i < 64; ++i){
    W[i] = sigma_1(W[i-2]) + W[i-7] + sigma_0(W[i-15]) + W[i-16];
  }

  uint32_t a = h[0];
  uint32_t b = h[1];
  uint32_t c = h[2];
  uint32_t d = h[3];
  uint32_t e = h[4];
  uint32_t f = h[5];
  uint32_t g = h[6];
  uint32_t htmp = h[7];

  for(int i = 0; i < 64; ++i){
    uint32_t T_1 = htmp + Sigma_1(e) + Ch(e,f,g) + constants[i] + W[i];
    uint32_t T_2 = Sigma_0(a) + Maj(a,b,c);
    htmp = g;
    g = f;
    f = e;
    e = d + T_1;
    d = c;
    c = b;
    b = a;
    a = T_1 + T_2;
  }

  h[0] += a;
  h[1] += b;
  h[2] += c;
  h[3] += d;
  h[4] += e;
  h[5] += f;
  h[6] += g;
  h[7] += htmp;

  hash[id * 8] = h[0];
  hash[id * 8 + 1] = h[1];
  hash[id * 8 + 2] = h[2];
  hash[id * 8 + 3] = h[3];
  hash[id * 8 + 4] = h[4];
  hash[id * 8 + 5] = h[5];
  hash[id * 8 + 6] = h[6];
  hash[id * 8 + 7] = h[7];
}

__global__ void found(uint32_t *hash, bool *found, uint32_t number_of_zeroes){
  int id = threadIdx.x + blockIdx.x * blockDim.x;

  uint32_t number_zerous_tmp = 0;

  if((hash[id * 8 + 7] & masks[0]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 7] & masks[1]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 7] & masks[2]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 7] & masks[3]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 7] & masks[4]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 7] & masks[5]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 7] & masks[6]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 7] & masks[7]) == 0)
    number_zerous_tmp++;

  if(number_of_zeroes == number_zerous_tmp){
    found[id] = true;
    return;
  } else if (number_of_zeroes < number_zerous_tmp){
    found[id] = false;
    return;
  } else if(number_zerous_tmp != 8){
    found[id] = false;
    return;
  }

  if((hash[id * 8 + 6] & masks[0]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 6] & masks[1]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 6] & masks[2]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 6] & masks[3]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 6] & masks[4]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 6] & masks[5]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 6] & masks[6]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 6] & masks[7]) == 0)
    number_zerous_tmp++;

  if(number_of_zeroes == number_zerous_tmp){
    found[id] = true;
    return;
  } else if (number_of_zeroes < number_zerous_tmp){
    found[id] = false;
    return;
  } else if(number_zerous_tmp != 16){
    found[id] = false;
    return;
  }

  if((hash[id * 8 + 5] & masks[0]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 5] & masks[1]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 5] & masks[2]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 5] & masks[3]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 5] & masks[4]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 5] & masks[5]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 5] & masks[6]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 5] & masks[7]) == 0)
    number_zerous_tmp++;

  if(number_of_zeroes == number_zerous_tmp){
    found[id] = true;
    return;
  } else if (number_of_zeroes < number_zerous_tmp){
    found[id] = false;
    return;
  } else if(number_zerous_tmp != 24){
    found[id] = false;
    return;
  }

  if((hash[id * 8 + 4] & masks[0]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 4] & masks[1]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 4] & masks[2]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 4] & masks[3]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 4] & masks[4]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 4] & masks[5]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 4] & masks[6]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 4] & masks[7]) == 0)
    number_zerous_tmp++;

  if(number_of_zeroes == number_zerous_tmp){
    found[id] = true;
    return;
  } else if (number_of_zeroes < number_zerous_tmp){
    found[id] = false;
    return;
  } else if(number_zerous_tmp != 32){
    found[id] = false;
    return;
  }

  if((hash[id * 8 + 3] & masks[0]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 3] & masks[1]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 3] & masks[2]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 3] & masks[3]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 3] & masks[4]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 3] & masks[5]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 3] & masks[6]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 3] & masks[7]) == 0)
    number_zerous_tmp++;

  if(number_of_zeroes == number_zerous_tmp){
    found[id] = true;
    return;
  } else if (number_of_zeroes < number_zerous_tmp){
    found[id] = false;
    return;
  } else if(number_zerous_tmp != 40){
    found[id] = false;
    return;
  }

  if((hash[id * 8 + 2] & masks[0]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 2] & masks[1]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 2] & masks[2]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 2] & masks[3]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 2] & masks[4]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 2] & masks[5]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 2] & masks[6]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 2] & masks[7]) == 0)
    number_zerous_tmp++;

  if(number_of_zeroes == number_zerous_tmp){
    found[id] = true;
    return;
  } else if (number_of_zeroes < number_zerous_tmp){
    found[id] = false;
    return;
  } else if(number_zerous_tmp != 48){
    found[id] = false;
    return;
  }

  if((hash[id * 8 + 1] & masks[0]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 1] & masks[1]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 1] & masks[2]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 1] & masks[3]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 1] & masks[4]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 1] & masks[5]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 1] & masks[6]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8 + 1] & masks[7]) == 0)
    number_zerous_tmp++;

  if(number_of_zeroes == number_zerous_tmp){
    found[id] = true;
    return;
  } else if (number_of_zeroes < number_zerous_tmp){
    found[id] = false;
    return;
  } else if(number_zerous_tmp != 56){
    found[id] = false;
    return;
  }

  if((hash[id * 8] & masks[0]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8] & masks[1]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8] & masks[2]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8] & masks[3]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8] & masks[4]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8] & masks[5]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8] & masks[6]) == 0)
    number_zerous_tmp++;

  if((hash[id * 8] & masks[7]) == 0)
    number_zerous_tmp++;

  if(number_of_zeroes == number_zerous_tmp){
    found[id] = true;
    return;
  } else {
    found[id] = false;
    return;
  }
}

int main() {
  const unsigned int threadsPerBlock = 16;
  const unsigned int blockCount = 15000;
  const unsigned int totalThreads = threadsPerBlock * blockCount;
  const uint32_t number_of_trailing_zeroes = 7;

  std::ifstream seeds("seeds.dat");
  unsigned long long int seed;
  bool found_flag = false;
  while(!seeds.eof()){
    seeds >> seed;
    hiprandState *devStates;
    hipMalloc((void **)&devStates, totalThreads * sizeof(hiprandState));
    setup_kernel<<<blockCount, threadsPerBlock>>>(devStates, seed);

    bool *found_host = new bool[totalThreads]();
    bool *found_device;
    uint32_t *m_host = new uint32_t[totalThreads * 2];
    uint32_t *m_device;
    uint32_t *hash_host = new uint32_t[totalThreads * 8];
    uint32_t *hash_device;
    hipMalloc(&hash_device, totalThreads * 8 * sizeof(uint32_t));
    hipMalloc(&m_device, totalThreads * 2 * sizeof(uint32_t));
    hipMalloc(&found_device, totalThreads * sizeof(bool));
    hipMemcpy(hash_device, hash_host, totalThreads * 8 * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(m_device, m_host, totalThreads * 2 * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(found_device, found_host, totalThreads * sizeof(bool), hipMemcpyHostToDevice);
    SHA_256_64<<<blockCount, threadsPerBlock>>>(devStates, hash_device, m_device);
    found<<<blockCount, threadsPerBlock>>>(hash_device, found_device, number_of_trailing_zeroes);
    hipDeviceSynchronize();
    hipMemcpy(hash_host, hash_device, totalThreads * 8 * sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipMemcpy(found_host, found_device, totalThreads * sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(m_host, m_device, totalThreads * 2 * sizeof(bool), hipMemcpyDeviceToHost);

    for(size_t index_hashes = 0; index_hashes < totalThreads; ++index_hashes){
      if(found_host[index_hashes]){
        found_flag = true;
        std::cout << "Message: " << std::hex << std::setw(8) << std::setfill('0') << m_host[index_hashes*2] << " ";
        std::cout << std::hex << std::setw(8) << std::setfill('0') << m_host[index_hashes*2 + 1]<< "\n";
        std::cout << "Hash: ";
        for(size_t index_in_hash = 0; index_in_hash < 8; ++index_in_hash){
          std::cout << std::hex << std::setw(8) << std::setfill('0') << std::hex << hash_host[index_hashes * 8 + index_in_hash] << " ";
        }
        std::cout << "\n";
        break;
      }
    }

    hipFree(hash_device);
    hipFree(found_device);
    hipFree(m_device);
    free(m_host);
    free(found_host);
    free(hash_host);
    if(found_flag)
      break;
  }
  return 0;
}
